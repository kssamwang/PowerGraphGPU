#include "hip/hip_runtime.h"
#include "LabelPropagationGPU_kernel.h"

__global__ void MSGApply_kernel(Vertex *vSet, LPA_Value *vValues, int numOfMsg, LPA_MSG *mValueSet)
{
	int tid = threadIdx.x;

	if(tid < numOfMsg)
	{
		LPA_MSG msg = mValueSet[tid];
		int destVId = msg.destVId;
		int index = msg.mValueIndex;
		int label = msg.label;

		vValues[index].label = label;
        vValues[index].destVId = destVId;
	}
}

hipError_t MSGApply_kernel_exec(Vertex *vSet, LPA_Value *vValues, int numOfMsg, LPA_MSG *mValueSet)
{
	hipError_t err = hipSuccess;
	
	MSGApply_kernel<<<1, NUMOFGPUCORE>>>(vSet, vValues, numOfMsg, mValueSet);
    err = hipGetLastError();

	hipDeviceSynchronize();
	
	return err;
}

__global__ void MSGGenMerge_kernel(LPA_MSG *mTransformdMergedMSGValueSet, Vertex *vSet, LPA_Value *vValues, int numOfEdge, Edge *eSet, int batchCnt)
{
	int tid = threadIdx.x;

	if(tid < numOfEdge)
	{
		int destVId = eSet[tid].dst;
		int srcVId = eSet[tid].src;
		int mValueIndex = (batchCnt << NUMOFGPUCORE_BIT) + tid;

		//test
//		printf("destvid %d srcvid %d label %d index %d\n", destVId, srcVId, vValues[srcVId].label, mValueIndex);

		mTransformdMergedMSGValueSet[mValueIndex].destVId = destVId;
		mTransformdMergedMSGValueSet[mValueIndex].label = vValues[srcVId].label;
	}
}

hipError_t MSGGenMerge_kernel_exec(LPA_MSG *mTransformdMergedMSGValueSet,
	Vertex *vSet, LPA_Value *vValues, int numOfEdge, Edge *eSet, int batchCnt)
{
	hipError_t err = hipSuccess;

	MSGGenMerge_kernel<<<1, NUMOFGPUCORE>>>(mTransformdMergedMSGValueSet, vSet, vValues, numOfEdge, eSet, batchCnt);
	err = hipGetLastError();

	hipDeviceSynchronize();
	
	return err;
}